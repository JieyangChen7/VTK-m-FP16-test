#include "hip/hip_runtime.h"
#include <vtkm/Types.h>
#include <vtkm/cont/DataSet.h>
#include <vtkm/cont/DataSetBuilderUniform.h>
#include <vtkm/worklet/Invoker.h>
#include <vtkm/cont/VariantArrayHandle.h>
#include <vtkm/worklet/WorkletMapField.h>
#include <vtkm/worklet/DispatcherMapField.h>
#include <vtkm/cont/DataSetFieldAdd.h>
#include <vtkm/filter/FilterField.h>
#include <vtkm/cont/ArrayHandle.h>
#include <iostream>
#include <vtkm/filter/internal/CreateResult.h> 
#include <vtkm/cont/Initialize.h>
#include <string>
#include <iomanip>
#include <stdlib.h> 
#include <vtkm/cont/Field.h>
#include "square_filter_fp16.h"
#include "square_filter_fp16_2.h"
#include "square_filter_fp32.h"
#include "square_filter_fp64.h"
#include <chrono> 
#define E 0.1


int main(int argc, char *argv[]) {
  vtkm::cont::InitializeOptions options =vtkm::cont::InitializeOptions::AddHelp;  
  vtkm::cont::Initialize(argc, argv, options);

  int n = atoi(argv[1]);

  vtkm::cont::DataSet inputDataSet, outputDataSet;
  vtkm::cont::DataSetBuilderUniform dataSetBuilder;
  vtkm::cont::DataSetFieldAdd dsf;

  vtkm::Id3 dims(n, n, n);
  vtkm::Id3 org(0, 0, 0);
  vtkm::Id3 spc(1, 1, 1);

  vtkm::Int64 N = n * n * n;

  std::vector<float> init_data(N);
  for (vtkm::Int64 i = 0; i < N; i++)
    init_data[i] = 1.0;//((float) rand() / (RAND_MAX)); 

#if defined FP16
  std::cout << "Running in FP16" << std::endl;
  std::vector<vtkm::Float16> data(N);
  vtkm::cont::ArrayHandle<vtkm::Float16> fieldData;
  vtkm::filter::FilterFieldSquareFP16 filter;
  vtkm::filter::PolicyFP16DataSet policy;
  vtkm::cont::ArrayHandle<vtkm::Float16> output;
  for (vtkm::Int64 i = 0; i < N; i++)
    data[i] = (float)init_data[i];
#elif defined FP16_2
  std::cout << "Running in FP16_2" << std::endl;
  std::vector<vtkm::Float16_2> data(N);
  vtkm::cont::ArrayHandle<vtkm::Float16_2> fieldData;
  vtkm::filter::FilterFieldSquareFP16_2 filter;
  vtkm::filter::PolicyFP16_2DataSet policy;
  vtkm::cont::ArrayHandle<vtkm::Float16_2> output;
  for (int i = 0; i < N; i++)
    data[i] = vtkm::Float16_2((float)init_data[i], (float)init_data[i]);
#elif defined FP32
  std::cout << "Running in FP32" << std::endl;
  std::vector<vtkm::Float32> data(N);
  vtkm::cont::ArrayHandle<vtkm::Float32> fieldData;
  vtkm::filter::FilterFieldSquareFP32 filter;
  vtkm::filter::PolicyFP32DataSet policy;
  vtkm::cont::ArrayHandle<vtkm::Float32> output;
  for (int i = 0; i < N; i++)
    data[i] = (float)init_data[i];
#elif defined FP64
  std::cout << "Running in FP64" << std::endl;
  std::vector<vtkm::Float64> data(N);
  vtkm::cont::ArrayHandle<vtkm::Float64> fieldData;
  vtkm::filter::FilterFieldSquareFP64 filter;
  vtkm::filter::PolicyFP64DataSet policy;
  vtkm::cont::ArrayHandle<vtkm::Float64> output;
  for (int i = 0; i < N; i++)
    data[i] = (double)init_data[i];
#endif

  fieldData = vtkm::cont::make_ArrayHandle(data);

  std::string fieldName = "test_field";
  inputDataSet = dataSetBuilder.Create(dims, org, spc);      
  dsf.AddPointField(inputDataSet, fieldName, fieldData);

  filter.SetActiveField(fieldName);
  auto start = std::chrono::high_resolution_clock::now();
  fieldData.PrepareForInPlace(vtkm::cont::DeviceAdapterTagCuda());
  auto finish = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = finish - start;
  std::cout << "Elapsed time: " << elapsed.count() << " s\n";
   
  outputDataSet = filter.Execute(inputDataSet, policy);
  outputDataSet = filter.Execute(inputDataSet, policy);
  outputDataSet = filter.Execute(inputDataSet, policy);
  
  //outputDataSet = fp16_filter.Execute(inputDataSet);
  start = std::chrono::high_resolution_clock::now();
  outputDataSet = filter.Execute(inputDataSet, policy);
  finish = std::chrono::high_resolution_clock::now();
  elapsed = finish - start;
  std::cout << "Elapsed time: " << elapsed.count() << " s\n";

  //std::cout << "Gflops: " << ((double)N/1e-9)/elapsed.count() << " s\n";
#if defined FP16_2
  std::cout << "Gflops: " << ((double)N*2/1e9)/elapsed.count() << " s\n";
#else
  std::cout << "Gflops: " << ((double)N/1e9)/elapsed.count() << "\n";
#endif
/*
  vtkm::cont::Field f = outputDataSet.GetField(fieldName+"_squared");
  vtkm::cont::VariantArrayHandle vah = f.GetData();
    
  vah.CopyTo(output);
  //std::cout << "Output: ";
  bool isCurrect = true;
  for (vtkm::Int64 i = 0; i < N; i++) {
    //std::cout << output.GetPortalConstControl().Get(i) << ", ";
#if defined FP16   
    if (fabs(i*i - output.GetPortalConstControl().Get(i).to_float()) > E)
#elif defined FP16_2
    if (fabs((float)i*i - output.GetPortalConstControl().Get(i).to_float_low()) > E ||
        fabs((float)i*i - output.GetPortalConstControl().Get(i).to_float_high()) > E)
#elif defined FP32
    if (fabs((float)i*i - output.GetPortalConstControl().Get(i)) > E)
#elif defined FP64
    if (fabs((double)i*i - output.GetPortalConstControl().Get(i)) > E)
#endif

    {
      std::cout << std::setprecision(10) << std::fixed;
      std::cout << "i = " << init_data[i] << std::endl;
      std::cout << "(float)i*i = " << (float)init_data[i]*init_data[i] << std::endl;
      std::cout << "PF = " << output.GetPortalConstControl().Get(i).to_float() << std::endl;
      isCurrect = false;
    }
  }
  if (isCurrect) std::cout << "Passed" << std::endl;
  else std::cout << "Failed" << std::endl;
  //std::cout << isCurrect ? "Passed":"Failed" << std::endl;
*/

}
